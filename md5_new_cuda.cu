#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <algorithm>

#define num_hashes 10000000
#define len 20
#define len_bit 160
#define OUT stdout

#define num_thread 64
#define num_block 384

unsigned char input[10000000];
unsigned int result[400000];

#define cs(a,b) ((a<<b) | (a>>(32-b)));
#define bswap(a) ((a>>24) | ((a<<8)&0xff0000) | ((a>>8)&0xff00) | (a<<24));

#define F(a, b, c) (c ^ (a & (b ^ c))) 
#define G(a, b, c) (((a ^ b) & c) ^ b)
#define H(a, b, c) (a ^ b ^ c)
#define I(a, b, c) (b ^ (a | (~c)))

#define A 0x67452301
#define B 0xefcdab89
#define C 0x98badcfe
#define D 0x10325476

__global__
void md5_gpu(unsigned char *input_gpu, unsigned int *result_gpu) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int m[16] = {0,};
	input_gpu[idx * 64 + len] = 0x80;
	input_gpu[idx * 64 + 56] = len_bit;
	memcpy(m, input_gpu, 64);
	
	a = B + cs((A + F(B, C, D) + m[ 0] + 0xd76aa478),  7);
	d = a + cs((D + F(a, B, C) + m[ 1] + 0xe8c7b756), 12);
	c = d + cs((C + F(d, a, B) + m[ 2] + 0x242070db), 17);
	b = c + cs((B + F(c, d, a) + m[ 3] + 0xc1bdceee), 22);
	a = b + cs((a + F(b, c, d) + m[ 4] + 0xf57c0faf),  7);
	d = a + cs((d + F(a, b, c) + m[ 5] + 0x4787c62a), 12);
	c = d + cs((c + F(d, a, b) + m[ 6] + 0xa8304613), 17);
	b = c + cs((b + F(c, d, a) /*+ m[ 7]*/ + 0xfd469501), 22);
	a = b + cs((a + F(b, c, d) /*+ m[ 8]*/ + 0x698098d8),  7);
	d = a + cs((d + F(a, b, c) /*+ m[ 9]*/ + 0x8b44f7af), 12);
	c = d + cs((c + F(d, a, b) /*+ m[10]*/ + 0xffff5bb1), 17);
	b = c + cs((b + F(c, d, a) /*+ m[11]*/ + 0x895cd7be), 22);
	a = b + cs((a + F(b, c, d) /*+ m[12]*/ + 0x6b901122),  7);
	d = a + cs((d + F(a, b, c) /*+ m[13]*/ + 0xfd987193), 12);
	c = d + cs((c + F(d, a, b) + m[14] + 0xa679438e), 17);
	b = c + cs((b + F(c, d, a) + m[15] + 0x49b40821), 22);
	
	a = b + cs((a + G(b, c, d) + m[ 1] + 0xf61e2562),  5);
	d = a + cs((d + G(a, b, c) + m[ 6] + 0xc040b340),  9);
	c = d + cs((c + G(d, a, b) /*+ m[11]*/ + 0x265e5a51), 14);
	b = c + cs((b + G(c, d, a) + m[ 0] + 0xe9b6c7aa), 20);
	a = b + cs((a + G(b, c, d) + m[ 5] + 0xd62f105d),  5);
	d = a + cs((d + G(a, b, c) /*+ m[10]*/ + 0x02441453),  9);
	c = d + cs((c + G(d, a, b) + m[15] + 0xd8a1e681), 14);
	b = c + cs((b + G(c, d, a) + m[ 4] + 0xe7d3fbc8), 20);
	a = b + cs((a + G(b, c, d) /*+ m[ 9]*/ + 0x21e1cde6),  5);
	d = a + cs((d + G(a, b, c) + m[14] + 0xc33707d6),  9);
	c = d + cs((c + G(d, a, b) + m[ 3] + 0xf4d50d87), 14);
	b = c + cs((b + G(c, d, a) /*+ m[ 8]*/ + 0x455a14ed), 20);
	a = b + cs((a + G(b, c, d) /*+ m[13]*/ + 0xa9e3e905),  5);
	d = a + cs((d + G(a, b, c) + m[ 2] + 0xfcefa3f8),  9);
	c = d + cs((c + G(d, a, b) /*+ m[ 7]*/ + 0x676f02d9), 14);
	b = c + cs((b + G(c, d, a) /*+ m[12]*/ + 0x8d2a4c8a), 20);
	
	a = b + cs((a + H(b, c, d) + m[ 5] + 0xfffa3942),  4);
	d = a + cs((d + H(a, b, c) /*+ m[ 8]*/ + 0x8771f681), 11);
	c = d + cs((c + H(d, a, b) /*+ m[11]*/ + 0x6d9d6122), 16);
	b = c + cs((b + H(c, d, a) + m[14] + 0xfde5380c), 23);
	a = b + cs((a + H(b, c, d) + m[ 1] + 0xa4beea44),  4);
	d = a + cs((d + H(a, b, c) + m[ 4] + 0x4bdecfa9), 11);
	c = d + cs((c + H(d, a, b) /*+ m[ 7]*/ + 0xf6bb4b60), 16);
	b = c + cs((b + H(c, d, a) /*+ m[10]*/ + 0xbebfbc70), 23);
	a = b + cs((a + H(b, c, d) /*+ m[13]*/ + 0x289b7ec6),  4);
	d = a + cs((d + H(a, b, c) + m[ 0] + 0xeaa127fa), 11);
	c = d + cs((c + H(d, a, b) + m[ 3] + 0xd4ef3085), 16);
	b = c + cs((b + H(c, d, a) + m[ 6] + 0x04881d05), 23);
	a = b + cs((a + H(b, c, d) /*+ m[ 9]*/ + 0xd9d4d039),  4);
	d = a + cs((d + H(a, b, c) /*+ m[12]*/ + 0xe6db99e5), 11);
	c = d + cs((c + H(d, a, b) + m[15] + 0x1fa27cf8), 16);
	b = c + cs((b + H(c, d, a) + m[ 2] + 0xc4ac5665), 23);
	
	a = b + cs((a + I(b, c, d) + m[ 0] + 0xf4292244),  6);
	d = a + cs((d + I(a, b, c) /*+ m[ 7]*/ + 0x432aff97), 10);
	c = d + cs((c + I(d, a, b) + m[14] + 0xab9423a7), 15);
	b = c + cs((b + I(c, d, a) + m[ 5] + 0xfc93a039), 21);
	a = b + cs((a + I(b, c, d) /*+ m[12]*/ + 0x655b59c3),  6);
	d = a + cs((d + I(a, b, c) + m[ 3] + 0x8f0ccc92), 10);
	c = d + cs((c + I(d, a, b) /*+ m[10]*/ + 0xffeff47d), 15);
	b = c + cs((b + I(c, d, a) + m[ 1] + 0x85845dd1), 21);
	a = b + cs((a + I(b, c, d) /*+ m[ 8]*/ + 0x6fa87e4f),  6);
	d = a + cs((d + I(a, b, c) + m[15] + 0xfe2ce6e0), 10);
	c = d + cs((c + I(d, a, b) + m[ 6] + 0xa3014314), 15);
	b = c + cs((b + I(c, d, a) /*+ m[13]*/ + 0x4e0811a1), 21);
	a = b + cs((a + I(b, c, d) + m[ 4] + 0xf7537e82),  6);
	d = a + cs((d + I(a, b, c) /*+ m[11]*/ + 0xbd3af235), 10);
	c = d + cs((c + I(d, a, b) + m[ 2] + 0x2ad7d2bb), 15);
	b = B + c + cs((b + I(c, d, a) /*+ m[ 9]*/ + 0xeb86d391), 21);
	result_gpu[4*idx+0] = bswap(a+A);
	result_gpu[4*idx+1] = bswap(b);
	result_gpu[4*idx+2] = bswap(c+C);
	result_gpu[4*idx+3] = bswap(d+D);
	result_gpu[0] = 4324234;
}

//max length: 55 bytes

int main(int argc, char *argv[]) {
	int workernum = atoi(argv[1]);
	char *inp = argv[2];

	char *basestr = inp; //"!\"#$%%&'()*+,-./0123456789:;<=>?@ABCDEFGHIJKLMNOPQRSTUVWXYZ[\\]^_`abcdefghijklmnopqrstuvwxyz{|}~";

	for (int i = 0; i < num_block; i++) {
		for (int j = 0; j < num_thread; j++) {
			input[(i*num_thread+j)*64] = basestr[i];
			input[(i*num_thread+j)*64+1] = basestr[j / 64];
			input[(i*num_thread+j)*64+2] = basestr[j % 64];
			for(int k = 3; k < len; k++) {
				input[(i*num_thread+j)*64+k] = inp[k];
			}
		}
	}

	long long int cnt = 0;
	time_t tstart = time(NULL);
	clock_t start;

	unsigned int Mr[4] = {0,};
	unsigned int mr[4] = {0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF};
	char Ms[100] = {0,};
	char ms[100] = {0,};

	int popcntM = 0;
	int popcntm = 128;
	char pMs[100] = {0,};
	char pms[100] = {0,};
	int popcnt_tmp;

	int base_tmp;
	unsigned int basem = 128;
	char bms[100] = {0,};

	unsigned int t_tmp;
	unsigned int tmax = 0;
	unsigned int tmin = 0x1000;
	char tMs[100] = {0,};
	char tms[100] = {0,};

	int SIZE_RESULT_GPU = 4 * num_block * num_thread;
	int SIZE_INPUT_GPU = 64 * num_block * num_thread;

	unsigned int *_result_gpu;
	unsigned char *_input_gpu;

	//while(1) {
		cnt++;
		start = clock();

		hipMalloc((void**)&_result_gpu, SIZE_RESULT_GPU);
		hipError_t err;
		if ((err = hipGetLastError()) != hipSuccess) {
			printf("%s\n", hipGetErrorString(err));
			return 0;
		}
		hipMalloc((void**)&_input_gpu, SIZE_INPUT_GPU);
		hipMemcpy(_input_gpu, input, SIZE_INPUT_GPU, hipMemcpyHostToDevice);
		md5_gpu<<<num_block, num_thread>>>(_input_gpu, _result_gpu);
		hipMemcpy(result, _result_gpu, SIZE_RESULT_GPU, hipMemcpyDeviceToHost);
		hipMemcpy(input, _input_gpu, SIZE_INPUT_GPU, hipMemcpyDeviceToHost);
		hipFree(_result_gpu);
		hipFree(_input_gpu);
		input[64] = 0;
		fprintf(OUT, "%d%d%d%d (%s)", result[0], result[1], result[2], result[3], input);
		//fprintf(OUT, "9||%.0lf||%.2lf||%lld||\n", num_hashes * 11.0 / (clock() - start) * CLOCKS_PER_SEC, (1.0 * len * num_hashes * 1.0 / (clock() - start) * CLOCKS_PER_SEC), cnt * num_hashes);
		fflush(OUT);
	//}
}